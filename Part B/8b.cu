
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) c[id] = a[id] + b[id];
}

int main( int argc, char* argv[] )
{
	int i, n = 100;
	double *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;

	size_t bytes = n * sizeof(double);

	h_a = (double*)malloc(bytes);
	h_b = (double*)malloc(bytes);
	h_c = (double*)malloc(bytes);
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);
	
	for(i = 0; i < n; i++) h_a[i] = h_b[i] = i;

	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	int blockSize, gridSize;
	blockSize = 1024;
	gridSize = (int)ceil((float)n/blockSize);

	vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	for(i=0; i<n; i++) printf(" %f + %f =%f\n",h_a[i],h_b[i],h_c[i]);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(h_a);
	free(h_b);
	free(h_c);
	return 0;
}
