#include "hip/hip_runtime.h"
%%cu

#include<stdio.h>
#define BLOCK_WIDTH 4
#define W 4

__global__ void matMul(double* a, double* b, double* c, int w)
{
    int row = threadIdx.x;
    int col = threadIdx.y;
    int value = 0;
    for(int i = 0; i < w; i++)
    {
        int a_var = a[col * w + i];
        int b_var = b[i * w + row];
        value += a_var * b_var;
    }
    c[col * w + row] = value;
}

int main()
{
    int i, size = W * W;
    double *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
 
    size_t bytes = size * sizeof(double);
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    for(i = 0; i < size; i++) h_a[i] = h_b[i] = i;
    
    printf("\n\nMatrix A:\n");
    for(i = 0; i < size; i++)
    {
        printf("%f\t", h_a[i]);
        if((i+1) % W == 0) printf("\n");
    }
 
    printf("\n\nMatrix B:\n");
    for(i = 0; i < size; i++)
    {
        printf("%f\t", h_b[i]);
        if((i+1) % W == 0) printf("\n");
    }
 
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    dim3 threads(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 grid(W / threads.x, W / threads.y);
 
    matMul<<<grid, threads>>>(d_a, d_b, d_c, W);
 
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
 
    printf("\n\nMatrix C:\n");
    for(i = 0; i < size; i++)
    {
        printf("%f\t", h_c[i]);
        if((i+1) % W == 0) printf("\n");
    }
 
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
