#include "hip/hip_runtime.h"
%%cu

#include <stdio.h>

__global__ void LM(float x)
{
	float f = x; 	 
}


__global__ void GM(float *array)
{
	array[threadIdx.x] = 2.0 * (float)threadIdx.x;
}


__global__ void SM(float *array)
{
	int i, index = threadIdx.x;
	float average, sum = 0.0;
 
  __shared__ float shared_arr[128];
	shared_arr[index] = array[index];
 
	__syncthreads();

  for(i = 0; i <= index; i++) sum += shared_arr[i];

  average = sum / (index + 1);
	printf("Thread id = %d\t Average = %f\n", index, average);
}

int main(int argc, char **argv)
{
  float h_arr[128], *d_arr;
  hipMalloc((void **) &d_arr, sizeof(float) * 128);
 
  LM<<<1, 128>>>(2.0);
 
  hipMemcpy((void *)d_arr, (void *)h_arr, sizeof(float) * 128, hipMemcpyHostToDevice);
  GM<<<1, 128>>>(d_arr);
  
  hipMemcpy((void *)h_arr, (void *)d_arr, sizeof(float) * 128, hipMemcpyDeviceToHost);
  SM<<<1, 128>>>(d_arr);

	hipDeviceSynchronize();
 
	return 0;
}
