#include "hip/hip_runtime.h"
%%cu

#include <stdio.h>

__global__ void LM(float x)
{
	float f = x; 	 
}


__global__ void GM(float *array)
{
	array[threadIdx.x] = 1.0 * (float)threadIdx.x;
}


__global__ void SM(float *array)
{
	int i, index = threadIdx.x;
	float average, sum = 0.0;
 
	 __shared__ float shared_arr[128];
	shared_arr[index] = array[index];
 
	__syncthreads();

  	for(i = 0; i <= index; i++) sum += shared_arr[i];

  	average = sum / (index + 1);
	printf("Thread id = %d\t Average = %f\n", index, average);
}

int main(int argc, char **argv)
{
  	float h_arr[128], *d_arr;
    	size_t bytes = 128 * sizeof(float);
	hipMalloc(&d_arr, bytes);
 
	LM<<<1, 128>>>(2.0);
 
	hipMemcpy(d_arr, h_arr, bn, hipMemcpyHostToDevice);
	GM<<<1, 128>>>(d_arr);
  
	hipMemcpy(h_arr, d_arr, sizeof(float) * 128, hipMemcpyDeviceToHost);
	SM<<<1, 128>>>(d_arr);

	hipDeviceSynchronize();
}
